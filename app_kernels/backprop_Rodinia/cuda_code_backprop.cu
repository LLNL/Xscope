
#include <hip/hip_runtime.h>

__device__
double  squash(double x0) {
  //float m;
  //x = -x;
  //m = 1 + x + x*x/2 + x*x*x/6 + x*x*x*x/24 + x*x*x*x*x/120;
  //return(1.0 / (1.0 + m));
  return (1.0 / (1.0 + exp(-x0)));
}

__global__ void kernel_1(
  double x0, double *ret) {
   *ret = squash(x0);
}

extern "C" {
double kernel_wrapper_1(double x0) {
  double *dev_p;
  hipMalloc(&dev_p, sizeof(double));
  kernel_1<<<1,1>>>(x0,dev_p);
  double res;
  hipMemcpy (&res, dev_p, sizeof(double), hipMemcpyDeviceToHost);
  return res;
  }
 }


