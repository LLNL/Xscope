
#include <hip/hip_runtime.h>

__device__
double ddot( unsigned n , const double *x , const double *y)
{
  double val = 0 ;
  const double * const x_end = x + n ;
  for ( ; x < x_end ; ++x , ++y ) { 
    val += *x * *y ; 
  }
  return val ;
}

__global__ void kernel_1(
  double x0, double x1, double *ret) {
   *ret = ddot(1, &x0, &x1);
}

extern "C" {
double kernel_wrapper_1(double x0, double x1) {
  double *dev_p;
  hipMalloc(&dev_p, sizeof(double));
  kernel_1<<<1,1>>>(x0,x1,dev_p);
  double res;
  hipMemcpy (&res, dev_p, sizeof(double), hipMemcpyDeviceToHost);
  return res;
  }
 }
